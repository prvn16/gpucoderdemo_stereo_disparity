#include "hip/hip_runtime.h"
/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * stereoDisparity.cu
 *
 * Code generation for function 'stereoDisparity'
 *
 */

/* Include files */
#include "rt_nonfinite.h"
#include "stereoDisparity.h"

/* Function Declarations */
static __global__ void stereoDisparity_kernel1(int16_T *out_disp, int32_T
  *min_cost);
static __global__ void stereoDisparity_kernel2(const uint8_T *img1, const
  uint8_T *img0, int32_T d, int32_T *diff_img);
static __global__ void stereoDisparity_kernel3(int32_T *diff_img, int32_T *a);
static __global__ void stereoDisparity_kernel4(int32_T *a, real_T *cost_v);
static __global__ void stereoDisparity_kernel5(real_T *cost_v, real_T *cost);
static __global__ void stereoDisparity_kernel6(int32_T d, real_T *cost, int16_T *
  out_disp, int32_T *min_cost);

/* Function Definitions */
static __global__ __launch_bounds__(512, 1) void stereoDisparity_kernel1(int16_T
  *out_disp, int32_T *min_cost)
{
  int32_T temp_cost;
  ;
  ;
  temp_cost = (int32_T)(((((gridDim.x * gridDim.y * blockIdx.z + gridDim.x *
    blockIdx.y) + blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z) +
    threadIdx.z * blockDim.x * blockDim.y) + threadIdx.y * blockDim.x) +
                        threadIdx.x);
  if (!(temp_cost >= 145408)) {
    /*  modified algorithm for stereo disparity block matching */
    /*  In this implementation instead of finding shifted image ,indices are mapped accordingly */
    /*  to save memory and some processing RGBA column major packed data is used as input for */
    /*  Compatibility with CUDA intrinsics Convolution is performed using separable filters (Horizontal and then Vertical) */
    /*  gpu code generation pragma */
    /*  Stereo disparity Parameters */
    /*  WIN_RAD is the radius of the window to be operated,min_disparity is the minimum disparity level  */
    /*  the search continues max_disparity is the maximun disparity level the search continues */
    /*  Image dimensions for loop control */
    /*  The number of channels packed are 4 (RGBA) so as nChannels are 4 */
    /*  To store the raw differences */
    /* To store the minimum cost */
    /*  Store the final disparity */
    min_cost[temp_cost] = 99999999;
    out_disp[temp_cost] = 0;
  }
}

static __global__ __launch_bounds__(512, 1) void stereoDisparity_kernel2(const
  uint8_T *img1, const uint8_T *img0, int32_T d, int32_T *diff_img)
{
  uint32_T threadId;
  int32_T ind_h;
  int32_T rowIdx;
  int32_T ind_w1;
  int32_T colIdx;
  int32_T ind_w2;
  int32_T tDiff;
  int32_T kk;
  int32_T temp_cost;
  ;
  ;
  threadId = ((((gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y) +
                blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z) +
               threadIdx.z * blockDim.x * blockDim.y) + threadIdx.y * blockDim.x)
    + threadIdx.x;
  colIdx = (int32_T)(threadId / 300U);
  rowIdx = (int32_T)(threadId - (uint32_T)colIdx * 300U);
  if ((!(rowIdx >= 300)) && (!(colIdx >= 528))) {
    /*  Row index calculation */
    ind_h = rowIdx - 7;

    /*  Column indices calculation for left image */
    ind_w1 = colIdx - 7;

    /*  Row indices calculation for right image */
    ind_w2 = (colIdx + d) - 23;

    /*  Border clamping for row Indices */
    if (rowIdx - 7 <= 0) {
      ind_h = 1;
    }

    if (ind_h > 284) {
      ind_h = 284;
    }

    /*  Border clamping for column indices for left image */
    if (colIdx - 7 <= 0) {
      ind_w1 = 1;
    }

    if (ind_w1 > 512) {
      ind_w1 = 512;
    }

    /*  Border clamping for column indices for right image */
    if (ind_w2 <= 0) {
      ind_w2 = 1;
    }

    if (ind_w2 > 512) {
      ind_w2 = 512;
    }

    /*  In this step, Sum of absolute Differences is performed */
    /*  across tour channels. */
    tDiff = 0;
    for (kk = 0; kk < 4; kk++) {
      temp_cost = (int32_T)img0[(((ind_h - 1) << 2) + kk) + 1136 * (ind_w1 - 1)]
        - (int32_T)img1[(((ind_h - 1) << 2) + kk) + 1136 * (ind_w2 - 1)];
      if (temp_cost < 0) {
        temp_cost = -temp_cost;
      }

      if ((tDiff < 0) && (temp_cost < MIN_int32_T - tDiff)) {
        tDiff = MIN_int32_T;
      } else if ((tDiff > 0) && (temp_cost > MAX_int32_T - tDiff)) {
        tDiff = MAX_int32_T;
      } else {
        tDiff += temp_cost;
      }
    }

    /* Store the SAD cost into a matrix */
    diff_img[rowIdx + 300 * colIdx] = tDiff;
  }
}

static __global__ __launch_bounds__(512, 1) void stereoDisparity_kernel3(int32_T
  *diff_img, int32_T *a)
{
  int32_T temp_cost;
  ;
  ;
  temp_cost = (int32_T)(((((gridDim.x * gridDim.y * blockIdx.z + gridDim.x *
    blockIdx.y) + blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z) +
    threadIdx.z * blockDim.x * blockDim.y) + threadIdx.y * blockDim.x) +
                        threadIdx.x);
  if (!(temp_cost >= 158400)) {
    /*  Aggregating the differences using separable convolution. Expect this to generate two Kernel */
    /*  using shared memory.The first kernel is the convolution with the horizontal kernel and second */
    /*  kernel operates on its output the column wise convolution. */
    a[temp_cost] = diff_img[temp_cost];
  }
}

static __global__ __launch_bounds__(1024, 1) void stereoDisparity_kernel4
  (int32_T *a, real_T *cost_v)
{
  real_T cv;
  int32_T temp_cost;
  int32_T threadIdY;
  int32_T threadIdX;
  __shared__ int32_T a_shared[1536];
  int32_T baseR;
  int32_T srow;
  int32_T strideRow;
  int32_T scol;
  int32_T strideCol;
  int32_T y_idx;
  int32_T baseC;
  int32_T x_idx;
  ;
  ;
  threadIdY = (int32_T)(blockDim.y * blockIdx.y + threadIdx.y);
  threadIdX = (int32_T)(blockDim.x * blockIdx.x + threadIdx.x);
  baseR = threadIdX;
  srow = (int32_T)threadIdx.x;
  strideRow = (int32_T)blockDim.x;
  scol = (int32_T)threadIdx.y;
  strideCol = (int32_T)blockDim.y;
  for (y_idx = srow; y_idx <= 31; y_idx += strideRow) {
    baseC = threadIdY;
    for (x_idx = scol; x_idx <= 47; x_idx += strideCol) {
      if ((baseR >= 0) && (baseR < 300) && ((baseC >= 0) && (baseC < 528))) {
        a_shared[y_idx + 32 * x_idx] = (int32_T)a[300 * baseC + baseR];
      } else {
        a_shared[y_idx + 32 * x_idx] = 0;
      }

      baseC += strideCol;
    }

    baseR += strideRow;
  }

  __syncthreads();
  if ((!(threadIdX >= 300)) && (!(threadIdY >= 512))) {
    cv = 0.0;
    for (temp_cost = 0; temp_cost < 17; temp_cost++) {
      cv += (real_T)a_shared[(int32_T)threadIdx.x + 32 * ((int32_T)threadIdx.y +
        ((temp_cost + threadIdY) - threadIdY))];
    }

    cost_v[threadIdX + 300 * threadIdY] = cv;
  }
}

static __global__ __launch_bounds__(1024, 1) void stereoDisparity_kernel5(real_T
  *cost_v, real_T *cost)
{
  real_T cv;
  int32_T temp_cost;
  int32_T threadIdY;
  int32_T threadIdX;
  __shared__ real_T cost_v_shared[1536];
  int32_T baseR;
  int32_T srow;
  int32_T strideRow;
  int32_T scol;
  int32_T strideCol;
  int32_T y_idx;
  int32_T baseC;
  int32_T x_idx;
  ;
  ;
  threadIdY = (int32_T)(blockDim.y * blockIdx.y + threadIdx.y);
  threadIdX = (int32_T)(blockDim.x * blockIdx.x + threadIdx.x);
  baseR = threadIdX;
  srow = (int32_T)threadIdx.x;
  strideRow = (int32_T)blockDim.x;
  scol = (int32_T)threadIdx.y;
  strideCol = (int32_T)blockDim.y;
  for (y_idx = srow; y_idx <= 47; y_idx += strideRow) {
    baseC = threadIdY;
    for (x_idx = scol; x_idx <= 31; x_idx += strideCol) {
      if ((baseR >= 0) && (baseR < 300) && ((baseC >= 0) && (baseC < 512))) {
        cost_v_shared[y_idx + 48 * x_idx] = (real_T)cost_v[300 * baseC + baseR];
      } else {
        cost_v_shared[y_idx + 48 * x_idx] = 0.0;
      }

      baseC += strideCol;
    }

    baseR += strideRow;
  }

  __syncthreads();
  if ((!(threadIdX >= 284)) && (!(threadIdY >= 512))) {
    cv = 0.0;
    for (temp_cost = 0; temp_cost < 17; temp_cost++) {
      cv += cost_v_shared[((int32_T)threadIdx.x + ((temp_cost + threadIdX) -
        threadIdX)) + 48 * (int32_T)threadIdx.y];
    }

    cost[threadIdX + 284 * threadIdY] = cv;
  }
}

static __global__ __launch_bounds__(512, 1) void stereoDisparity_kernel6(int32_T
  d, real_T *cost, int16_T *out_disp, int32_T *min_cost)
{
  uint32_T threadId;
  real_T cv;
  int32_T kk;
  int32_T colIdx;
  int32_T temp_cost;
  ;
  ;
  threadId = ((((gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y) +
                blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z) +
               threadIdx.z * blockDim.x * blockDim.y) + threadIdx.y * blockDim.x)
    + threadIdx.x;
  colIdx = (int32_T)(threadId / 284U);
  kk = (int32_T)(threadId - (uint32_T)colIdx * 284U);
  if ((!(kk >= 284)) && (!(colIdx >= 512))) {
    /*  load the cost */
    cv = cost[kk + 284 * colIdx];
    if (cv < 2.147483648E+9) {
      if (cv >= -2.147483648E+9) {
        temp_cost = (int32_T)cv;
      } else {
        temp_cost = MIN_int32_T;
      }
    } else if (cv >= 2.147483648E+9) {
      temp_cost = MAX_int32_T;
    } else {
      temp_cost = 0;
    }

    /*  compare against the minimum cost available and store the */
    /*  disparity value */
    if (min_cost[kk + 284 * colIdx] > temp_cost) {
      min_cost[kk + 284 * colIdx] = temp_cost;
      out_disp[kk + 284 * colIdx] = (int16_T)((int32_T)fabs(-16.0 + (real_T)d) +
        8);
    }
  }
}

void stereoDisparity(const uint8_T img0[581632], const uint8_T img1[581632],
                     int16_T out_disp[145408])
{
  int32_T d;
  int16_T *gpu_out_disp;
  int32_T *gpu_min_cost;
  uint8_T *gpu_img1;
  uint8_T *gpu_img0;
  int32_T *gpu_diff_img;
  int32_T *gpu_a;
  real_T *gpu_cost_v;
  real_T *gpu_cost;
  boolean_T img1_dirtyOnCpu;
  boolean_T img0_dirtyOnCpu;
  hipMalloc(&gpu_min_cost, 581632ULL);
  hipMalloc(&gpu_out_disp, 290816ULL);
  hipMalloc(&gpu_cost, 1163264ULL);
  hipMalloc(&gpu_cost_v, 1228800ULL);
  hipMalloc(&gpu_a, 633600ULL);
  hipMalloc(&gpu_diff_img, 633600ULL);
  hipMalloc(&gpu_img0, 581632ULL);
  hipMalloc(&gpu_img1, 581632ULL);
  img1_dirtyOnCpu = true;
  img0_dirtyOnCpu = true;

  /*  modified algorithm for stereo disparity block matching */
  /*  In this implementation instead of finding shifted image ,indices are mapped accordingly */
  /*  to save memory and some processing RGBA column major packed data is used as input for */
  /*  Compatibility with CUDA intrinsics Convolution is performed using separable filters (Horizontal and then Vertical) */
  /*  gpu code generation pragma */
  /*  Stereo disparity Parameters */
  /*  WIN_RAD is the radius of the window to be operated,min_disparity is the minimum disparity level  */
  /*  the search continues max_disparity is the maximun disparity level the search continues */
  /*  Image dimensions for loop control */
  /*  The number of channels packed are 4 (RGBA) so as nChannels are 4 */
  /*  To store the raw differences */
  /* To store the minimum cost */
  /*  Store the final disparity */
  stereoDisparity_kernel1<<<dim3(284U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (gpu_out_disp, gpu_min_cost);

  /*  Filters for aggregating the differences */
  /*  filter_h is the horizontal filter used in separable convolution */
  /*  filter_v is the vertical filter used in separable convolution which */
  /*  operates on the output of the row convolution */
  /*  Main Loop that runs for all the disparity levels. This loop is */
  /*  expected to run on CPU. */
  for (d = 0; d < 17; d++) {
    /*  Find the difference matrix for the current disparity level. Expect */
    /*  this to generate a Kernel function. */
    if (img1_dirtyOnCpu) {
      hipMemcpy((void *)gpu_img1, (void *)&img1[0], 581632ULL,
                 hipMemcpyHostToDevice);
      img1_dirtyOnCpu = false;
    }

    if (img0_dirtyOnCpu) {
      hipMemcpy((void *)gpu_img0, (void *)&img0[0], 581632ULL,
                 hipMemcpyHostToDevice);
      img0_dirtyOnCpu = false;
    }

    stereoDisparity_kernel2<<<dim3(310U, 1U, 1U), dim3(512U, 1U, 1U)>>>(gpu_img1,
      gpu_img0, d, gpu_diff_img);

    /*  Aggregating the differences using separable convolution. Expect this to generate two Kernel */
    /*  using shared memory.The first kernel is the convolution with the horizontal kernel and second */
    /*  kernel operates on its output the column wise convolution. */
    stereoDisparity_kernel3<<<dim3(310U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (gpu_diff_img, gpu_a);
    stereoDisparity_kernel4<<<dim3(10U, 16U, 1U), dim3(32U, 32U, 1U)>>>(gpu_a,
      gpu_cost_v);
    stereoDisparity_kernel5<<<dim3(9U, 16U, 1U), dim3(32U, 32U, 1U)>>>
      (gpu_cost_v, gpu_cost);

    /*  This part updates the min_cost matrix with by comparing the values */
    /*  with current disparity level. */
    stereoDisparity_kernel6<<<dim3(284U, 1U, 1U), dim3(512U, 1U, 1U)>>>(d,
      gpu_cost, gpu_out_disp, gpu_min_cost);
  }

  hipMemcpy((void *)&out_disp[0], (void *)gpu_out_disp, 290816ULL,
             hipMemcpyDeviceToHost);
  hipFree(gpu_img1);
  hipFree(gpu_img0);
  hipFree(gpu_diff_img);
  hipFree(gpu_a);
  hipFree(gpu_cost_v);
  hipFree(gpu_cost);
  hipFree(gpu_out_disp);
  hipFree(gpu_min_cost);
}

/* End of code generation (stereoDisparity.cu) */
