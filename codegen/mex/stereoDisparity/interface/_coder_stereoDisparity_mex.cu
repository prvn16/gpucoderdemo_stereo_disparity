/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * _coder_stereoDisparity_mex.cu
 *
 * Code generation for function '_coder_stereoDisparity_mex'
 *
 */

/* Include files */
#include "stereoDisparity.h"
#include "_coder_stereoDisparity_mex.h"
#include "stereoDisparity_terminate.h"
#include "_coder_stereoDisparity_api.h"
#include "stereoDisparity_initialize.h"
#include "stereoDisparity_data.h"

/* Function Declarations */
static void stereoDisparity_mexFunction(int32_T nlhs, mxArray *plhs[1], int32_T
  nrhs, const mxArray *prhs[2]);

/* Function Definitions */
static void stereoDisparity_mexFunction(int32_T nlhs, mxArray *plhs[1], int32_T
  nrhs, const mxArray *prhs[2])
{
  int32_T n;
  const mxArray *inputs[2];
  const mxArray *outputs[1];
  int32_T b_nlhs;

  /* Check for proper number of arguments. */
  if (nrhs != 2) {
    emlrtErrMsgIdAndTxt(emlrtRootTLSGlobal, "EMLRT:runTime:WrongNumberOfInputs",
                        5, 12, 2, 4, 15, "stereoDisparity");
  }

  if (nlhs > 1) {
    emlrtErrMsgIdAndTxt(emlrtRootTLSGlobal,
                        "EMLRT:runTime:TooManyOutputArguments", 3, 4, 15,
                        "stereoDisparity");
  }

  /* Temporary copy for mex inputs. */
  for (n = 0; n < nrhs; n++) {
    inputs[n] = prhs[n];
  }

  /* Call the function. */
  stereoDisparity_api(inputs, outputs);

  /* Copy over outputs to the caller. */
  if (nlhs < 1) {
    b_nlhs = 1;
  } else {
    b_nlhs = nlhs;
  }

  emlrtReturnArrays(b_nlhs, plhs, outputs);

  /* Module termination. */
  stereoDisparity_terminate();
}

void mexFunction(int32_T nlhs, mxArray *plhs[], int32_T nrhs, const mxArray
                 *prhs[])
{
  mexAtExit(stereoDisparity_atexit);

  /* Initialize the memory manager. */
  /* Module initialization. */
  stereoDisparity_initialize();

  /* Dispatch the entry-point. */
  stereoDisparity_mexFunction(nlhs, plhs, nrhs, prhs);
}

emlrtCTX mexFunctionCreateRootTLS(void)
{
  emlrtCreateRootTLS(&emlrtRootTLSGlobal, &emlrtContextGlobal, NULL, 1);
  return emlrtRootTLSGlobal;
}

/* End of code generation (_coder_stereoDisparity_mex.cu) */
