/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * _coder_stereoDisparity_api.cu
 *
 * Code generation for function '_coder_stereoDisparity_api'
 *
 */

/* Include files */
#include "rt_nonfinite.h"
#include "stereoDisparity.h"
#include "_coder_stereoDisparity_api.h"
#include "stereoDisparity_data.h"

/* Function Declarations */
static uint8_T (*b_emlrt_marshallIn(const mxArray *u, const emlrtMsgIdentifier
  *parentId))[581632];
static uint8_T (*c_emlrt_marshallIn(const mxArray *src, const emlrtMsgIdentifier
  *msgId))[581632];
static uint8_T (*emlrt_marshallIn(const mxArray *img0, const char_T *identifier))
  [581632];
static const mxArray *emlrt_marshallOut(const int16_T u[145408]);

/* Function Definitions */
static uint8_T (*b_emlrt_marshallIn(const mxArray *u, const emlrtMsgIdentifier
  *parentId))[581632]
{
  uint8_T (*y)[581632];
  y = c_emlrt_marshallIn(emlrtAlias(u), parentId);
  emlrtDestroyArray(&u);
  return y;
}
  static uint8_T (*c_emlrt_marshallIn(const mxArray *src, const
  emlrtMsgIdentifier *msgId))[581632]
{
  uint8_T (*ret)[581632];
  static const int32_T dims[2] = { 1136, 512 };

  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src, "uint8", false, 2U,
    dims);
  ret = (uint8_T (*)[581632])emlrtMxGetData(src);
  emlrtDestroyArray(&src);
  return ret;
}

static uint8_T (*emlrt_marshallIn(const mxArray *img0, const char_T *identifier))
  [581632]
{
  uint8_T (*y)[581632];
  emlrtMsgIdentifier thisId;
  thisId.fIdentifier = (const char *)identifier;
  thisId.fParent = NULL;
  thisId.bParentIsCell = false;
  y = b_emlrt_marshallIn(emlrtAlias(img0), &thisId);
  emlrtDestroyArray(&img0);
  return y;
}
  static const mxArray *emlrt_marshallOut(const int16_T u[145408])
{
  const mxArray *y;
  const mxArray *m0;
  static const int32_T iv0[2] = { 0, 0 };

  static const int32_T iv1[2] = { 284, 512 };

  y = NULL;
  m0 = emlrtCreateNumericArray(2, iv0, mxINT16_CLASS, mxREAL);
  emlrtMxSetData((mxArray *)m0, (void *)&u[0]);
  emlrtSetDimensions((mxArray *)m0, iv1, 2);
  emlrtAssign(&y, m0);
  return y;
}

void stereoDisparity_api(const mxArray * const prhs[2], const mxArray *plhs[1])
{
  int16_T (*out_disp)[145408];
  uint8_T (*img0)[581632];
  uint8_T (*img1)[581632];
  out_disp = (int16_T (*)[145408])mxMalloc(sizeof(int16_T [145408]));

  /* Marshall function inputs */
  img0 = emlrt_marshallIn(emlrtAlias(prhs[0]), "img0");
  img1 = emlrt_marshallIn(emlrtAlias(prhs[1]), "img1");

  /* Invoke the target function */
  stereoDisparity(*img0, *img1, *out_disp);

  /* Marshall function outputs */
  plhs[0] = emlrt_marshallOut(*out_disp);
}

/* End of code generation (_coder_stereoDisparity_api.cu) */
