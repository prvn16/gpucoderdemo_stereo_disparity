/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * stereoDisparity_terminate.cu
 *
 * Code generation for function 'stereoDisparity_terminate'
 *
 */

/* Include files */
#include "rt_nonfinite.h"
#include "stereoDisparity.h"
#include "stereoDisparity_terminate.h"
#include "_coder_stereoDisparity_mex.h"
#include "stereoDisparity_data.h"

/* Function Definitions */
void stereoDisparity_atexit(void)
{
  mexFunctionCreateRootTLS();
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
}

void stereoDisparity_terminate(void)
{
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
}

/* End of code generation (stereoDisparity_terminate.cu) */
