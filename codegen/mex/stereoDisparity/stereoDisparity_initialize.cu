/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * stereoDisparity_initialize.cu
 *
 * Code generation for function 'stereoDisparity_initialize'
 *
 */

/* Include files */
#include "rt_nonfinite.h"
#include "stereoDisparity.h"
#include "stereoDisparity_initialize.h"
#include "_coder_stereoDisparity_mex.h"
#include "stereoDisparity_data.h"

/* Function Definitions */
void stereoDisparity_initialize(void)
{
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, 0);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtFirstTimeR2012b(emlrtRootTLSGlobal);
}

/* End of code generation (stereoDisparity_initialize.cu) */
